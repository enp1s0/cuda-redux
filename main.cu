
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void redux_kernel(
  std::uint32_t* const out_ptr,
  const std::uint32_t* const in_ptr
  ) {
  const auto in = in_ptr[threadIdx.x];

  constexpr unsigned member_mask = ~0u;

  std::uint32_t out = 0;

  asm(
    R"(
{redux.sync.add.u32 %0, %1, %2;}
)":"=r"(out) : "r"(in), "r"(member_mask)
    );

  out_ptr[threadIdx.x] = out;
}

int main() {
  constexpr std::uint32_t warp_size = 32;

  std::uint32_t *in_ptr, *out_ptr;
  hipMallocManaged(&in_ptr , sizeof(std::uint32_t) * warp_size);
  hipMallocManaged(&out_ptr, sizeof(std::uint32_t) * warp_size);

  std::uint32_t ref = 0;
  for (unsigned i = 0; i < warp_size; i++) {
    ref += (in_ptr[i] = i);
  }

  redux_kernel<<<1, warp_size>>>(out_ptr, in_ptr);

  hipDeviceSynchronize();

  bool ok = true;
  for (unsigned i = 0; i < warp_size; i++) {
    if (out_ptr[i] != ref) {
      std::printf("out[%3u] = %u, ref = %u\n", i, out_ptr[i], ref);
      ok = false;
    }
  }
  std::printf("[result] %s\n", ok ? "ok" : "ng");

  hipFree(in_ptr);
  hipFree(out_ptr);
}
